/****************************************************************************
 * gemm_bf16_tf32.cu
 *   Compare GEMM performance among float32, tensorfloat32, and bfloat16
 *
 *   - float32       : normal FP32 GEMM
 *   - tensorfloat32 : set cublas TF32 math mode (still uses float* buffers)
 *   - bfloat16      : use __nv_bfloat16 buffers, cublasGemmEx with CUDA_R_16BF
 *
 * Build:
 *   nvcc -std=c++17 -o gemm_bf16_tf32 gemm_bf16_tf32.cu -lcublas
 ****************************************************************************/
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <random>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_bf16.h> // __nv_bfloat16, float2bfloat16, etc.

//---------------------------------------------------------
// Error-check macros
//---------------------------------------------------------
static void checkCuda(hipError_t err, const char* file, int line){
    if(err!=hipSuccess){
        std::cerr << "CUDA Error: " << hipGetErrorString(err)
                  << " at " << file << ":" << line << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK_CUDA(x) checkCuda(x, __FILE__, __LINE__)

static void checkCublas(hipblasStatus_t stat, const char* file, int line){
    if(stat!=HIPBLAS_STATUS_SUCCESS){
        std::cerr << "cuBLAS Error: code=" << (int)stat
                  << " at " << file << ":" << line << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK_CUBLAS(x) checkCublas(x, __FILE__, __LINE__)

//---------------------------------------------------------
// Device kernel: row->col (float)
//   in:  row-major [M*K]
//   out: col-major [M*K]
//   => out[col*M + row] = in[row*K + col]
//---------------------------------------------------------
__global__
void row2col_f32(const float* __restrict__ in,
                 float* __restrict__ out,
                 int M, int K)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < M*K){
        int row = tid / K;
        int col = tid % K;
        out[col*M + row] = in[tid];
    }
}

//---------------------------------------------------------
// Device kernel: col->row (float)
//   in:  col-major [M*K]
//   out: row-major [M*K]
//---------------------------------------------------------
__global__
void col2row_f32(const float* __restrict__ in,
                 float* __restrict__ out,
                 int M, int K)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < M*K){
        int row = tid / K;
        int col = tid % K;
        out[tid] = in[col*M + row];
    }
}

//---------------------------------------------------------
// Device kernel: row->col (bf16)
//---------------------------------------------------------
__global__
void row2col_bf16(const __hip_bfloat16* __restrict__ in,
                  __hip_bfloat16* __restrict__ out,
                  int M, int K)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < M*K){
        int row = tid / K;
        int col = tid % K;
        out[col*M + row] = in[tid];
    }
}

//---------------------------------------------------------
// Device kernel: col->row (bf16)
//---------------------------------------------------------
__global__
void col2row_bf16(const __hip_bfloat16* __restrict__ in,
                  __hip_bfloat16* __restrict__ out,
                  int M, int K)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < M*K){
        int row = tid / K;
        int col = tid % K;
        out[tid] = in[col*M + row];
    }
}

//---------------------------------------------------------
// Utility: compute GFLOPS = 2*M*N*K / (time_sec * 1e9)
//   ms: time in milliseconds
//---------------------------------------------------------
static float calcGflops(int M, int N, int K, float ms){
    double t_sec = ms * 1e-3;
    double flops = 2.0 * (double)M * (double)N * (double)K;
    double gflops = flops / (t_sec * 1e9);
    return (float)gflops;
}

//---------------------------------------------------------
// Run GEMM in float32
//   - A, B, C: row-major
//   - cublas expects col-major => device-side transpose
//---------------------------------------------------------
float gemm_fp32(hipblasHandle_t handle,
                int M, int N, int K,
                const float* dA_row, 
                const float* dB_row,
                float* dC_row,
                int repeat=10)
{
    // (1) Allocate col-major buffers
    float *dA_col=nullptr, *dB_col=nullptr, *dC_col=nullptr;
    CHECK_CUDA( hipMalloc(&dA_col, sizeof(float)*M*K) );
    CHECK_CUDA( hipMalloc(&dB_col, sizeof(float)*K*N) );
    CHECK_CUDA( hipMalloc(&dC_col, sizeof(float)*M*N) );

    // (2) row->col
    {
        dim3 block(256);
        dim3 gridA((M*K + block.x -1)/block.x);
        row2col_f32<<<gridA, block>>>(dA_row, dA_col, M, K);

        dim3 gridB((K*N + block.x -1)/block.x);
        row2col_f32<<<gridB, block>>>(dB_row, dB_col, K, N);
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // (3) Warm-up
    float alpha=1.0f, beta=0.0f;
    CHECK_CUBLAS( hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,  // no transpose
        M, N, K,
        &alpha,
        dA_col, HIP_R_32F, M,
        dB_col, HIP_R_32F, K,
        &beta,
        dC_col, HIP_R_32F, M,
        HIP_R_32F,                // computeType
        HIPBLAS_GEMM_DEFAULT        // algo
    ));
    CHECK_CUDA(hipDeviceSynchronize());

    // (4) Measure
    hipEvent_t startEv, stopEv;
    CHECK_CUDA(hipEventCreate(&startEv));
    CHECK_CUDA(hipEventCreate(&stopEv));
    CHECK_CUDA(hipEventRecord(startEv));

    for(int i=0; i<repeat; i++){
        CHECK_CUBLAS( hipblasGemmEx(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            M, N, K,
            &alpha,
            dA_col, HIP_R_32F, M,
            dB_col, HIP_R_32F, K,
            &beta,
            dC_col, HIP_R_32F, M,
            HIP_R_32F,
            HIPBLAS_GEMM_DEFAULT
        ));
    }
    CHECK_CUDA(hipEventRecord(stopEv));
    CHECK_CUDA(hipEventSynchronize(stopEv));
    float ms;
    CHECK_CUDA(hipEventElapsedTime(&ms, startEv, stopEv));
    ms /= repeat;

    // (5) col->row
    {
        dim3 block(256);
        dim3 gridC((M*N + block.x -1)/block.x);
        col2row_f32<<<gridC, block>>>(dC_col, dC_row, M, N);
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // cleanup
    CHECK_CUDA(hipFree(dA_col));
    CHECK_CUDA(hipFree(dB_col));
    CHECK_CUDA(hipFree(dC_col));
    CHECK_CUDA(hipEventDestroy(startEv));
    CHECK_CUDA(hipEventDestroy(stopEv));
    return ms;
}

//---------------------------------------------------------
// Run GEMM in tensorfloat32
//   - Almost same as gemm_fp32, but we enable TF32 math mode.
//   - Data buffers are still float*, but internally uses TF32
//---------------------------------------------------------
float gemm_tf32(hipblasHandle_t handle,
                int M, int N, int K,
                const float* dA_row,
                const float* dB_row,
                float* dC_row,
                int repeat=10)
{
    // Set TF32 math mode
    CHECK_CUBLAS( hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH) );

    // (1) Allocate col-major buffers
    float *dA_col=nullptr, *dB_col=nullptr, *dC_col=nullptr;
    CHECK_CUDA( hipMalloc(&dA_col, sizeof(float)*M*K) );
    CHECK_CUDA( hipMalloc(&dB_col, sizeof(float)*K*N) );
    CHECK_CUDA( hipMalloc(&dC_col, sizeof(float)*M*N) );

    // (2) row->col
    {
        dim3 block(256);
        dim3 gridA((M*K + block.x -1)/block.x);
        row2col_f32<<<gridA, block>>>(dA_row, dA_col, M, K);

        dim3 gridB((K*N + block.x -1)/block.x);
        row2col_f32<<<gridB, block>>>(dB_row, dB_col, K, N);
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // (3) Warm-up
    float alpha=1.0f, beta=0.0f;
    CHECK_CUBLAS( hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        M, N, K,
        &alpha,
        dA_col, HIP_R_32F, M,
        dB_col, HIP_R_32F, K,
        &beta,
        dC_col, HIP_R_32F, M,
        // computeType = CUDA_R_32F, but cublasSetMathMode(TF32) is active
        HIP_R_32F,
        CUBLAS_GEMM_ALGO0_TENSOR_OP
    ));
    CHECK_CUDA(hipDeviceSynchronize());

    // (4) Measure
    hipEvent_t startEv, stopEv;
    CHECK_CUDA(hipEventCreate(&startEv));
    CHECK_CUDA(hipEventCreate(&stopEv));
    CHECK_CUDA(hipEventRecord(startEv));

    for(int i=0; i<repeat; i++){
        CHECK_CUBLAS( hipblasGemmEx(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            M, N, K,
            &alpha,
            dA_col, HIP_R_32F, M,
            dB_col, HIP_R_32F, K,
            &beta,
            dC_col, HIP_R_32F, M,
            HIP_R_32F,
            CUBLAS_GEMM_DEFAULT_TENSOR_OP
        ));
    }
    CHECK_CUDA(hipEventRecord(stopEv));
    CHECK_CUDA(hipEventSynchronize(stopEv));
    float ms;
    CHECK_CUDA(hipEventElapsedTime(&ms, startEv, stopEv));
    ms /= repeat;

    // (5) col->row
    {
        dim3 block(256);
        dim3 gridC((M*N + block.x -1)/block.x);
        col2row_f32<<<gridC, block>>>(dC_col, dC_row, M, N);
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // restore math mode if needed
    CHECK_CUBLAS( hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH) );

    // cleanup
    CHECK_CUDA(hipFree(dA_col));
    CHECK_CUDA(hipFree(dB_col));
    CHECK_CUDA(hipFree(dC_col));
    CHECK_CUDA(hipEventDestroy(startEv));
    CHECK_CUDA(hipEventDestroy(stopEv));

    return ms;
}

//---------------------------------------------------------
// Run GEMM in BF16
//   - A, B, C are stored in __nv_bfloat16
//   - compute type is FP32
//---------------------------------------------------------
float gemm_bf16(hipblasHandle_t handle,
                int M, int N, int K,
                const __hip_bfloat16* dA_row,
                const __hip_bfloat16* dB_row,
                __hip_bfloat16* dC_row,
                int repeat=10)
{
    // (1) Allocate col-major buffers
    __hip_bfloat16 *dA_col=nullptr, *dB_col=nullptr, *dC_col=nullptr;
    CHECK_CUDA( hipMalloc(&dA_col, sizeof(__hip_bfloat16)*M*K) );
    CHECK_CUDA( hipMalloc(&dB_col, sizeof(__hip_bfloat16)*K*N) );
    CHECK_CUDA( hipMalloc(&dC_col, sizeof(__hip_bfloat16)*M*N) );

    // (2) row->col
    {
        dim3 block(256);
        dim3 gridA((M*K + block.x -1)/block.x);
        row2col_bf16<<<gridA, block>>>(dA_row, dA_col, M, K);

        dim3 gridB((K*N + block.x -1)/block.x);
        row2col_bf16<<<gridB, block>>>(dB_row, dB_col, K, N);
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // (3) Warm-up
    float alpha=1.0f, beta=0.0f;
    CHECK_CUBLAS( hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        M, N, K,
        &alpha,
        dA_col, HIP_R_16BF, M,  // dataType = BF16
        dB_col, HIP_R_16BF, K,
        &beta,
        dC_col, HIP_R_16BF, M,
        // computeType = CUBLAS_COMPUTE_32F (== CUDA_R_32F),
        //   but API expects "cudaDataType_t" => use CUDA_R_32F
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP

    ));
    CHECK_CUDA(hipDeviceSynchronize());

    // (4) Measure
    hipEvent_t startEv, stopEv;
    CHECK_CUDA(hipEventCreate(&startEv));
    CHECK_CUDA(hipEventCreate(&stopEv));
    CHECK_CUDA(hipEventRecord(startEv));

    for(int i=0; i<repeat; i++){
        CHECK_CUBLAS( hipblasGemmEx(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            M, N, K,
            &alpha,
            dA_col, HIP_R_16BF, M,
            dB_col, HIP_R_16BF, K,
            &beta,
            dC_col, HIP_R_16BF, M,
            HIP_R_32F,
            CUBLAS_GEMM_DEFAULT_TENSOR_OP
        ));
    }
    CHECK_CUDA(hipEventRecord(stopEv));
    CHECK_CUDA(hipEventSynchronize(stopEv));
    float ms;
    CHECK_CUDA(hipEventElapsedTime(&ms, startEv, stopEv));
    ms /= repeat;

    // (5) col->row
    {
        dim3 block(256);
        dim3 gridC((M*N + block.x -1)/block.x);
        col2row_bf16<<<gridC, block>>>(dC_col, dC_row, M, N);
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // cleanup
    CHECK_CUDA(hipFree(dA_col));
    CHECK_CUDA(hipFree(dB_col));
    CHECK_CUDA(hipFree(dC_col));
    CHECK_CUDA(hipEventDestroy(startEv));
    CHECK_CUDA(hipEventDestroy(stopEv));
    return ms;
}

//---------------------------------------------------------
// main
//---------------------------------------------------------
int main()
{
    // create cublas handle
    hipblasHandle_t handle;
    CHECK_CUBLAS( hipblasCreate(&handle) );

    // Generate random M, N, K
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> dist(128, 512);
    int M = dist(gen);
    int K = dist(gen);
    int N = dist(gen);

    // For demonstration, we pick M,K,N in multiples of 8 to align well
    // but it's optional
    M = M * 64;
    K = K * 64;
    N = N * 64;
    if(M<64) M=64;
    if(K<64) K=64;
    if(N<64) N=64;

    std::cout << "GEMM size: M=" << M << ", K=" << K << ", N=" << N << "\n";

    // allocate host memory
    std::vector<float> hA_fp32(M*K), hB_fp32(K*N), hC_fp32(M*N, 0.f);

    // random initialization
    {
        std::mt19937 rg(1234);
        std::uniform_real_distribution<float> distf(0.f, 1.f);
        for(int i=0; i<M*K; i++){
            hA_fp32[i] = distf(rg);
        }
        for(int i=0; i<K*N; i++){
            hB_fp32[i] = distf(rg);
        }
    }

    // device memory for float32
    float *dA_fp32=nullptr, *dB_fp32=nullptr, *dC_fp32=nullptr;
    CHECK_CUDA( hipMalloc(&dA_fp32, sizeof(float)*M*K) );
    CHECK_CUDA( hipMalloc(&dB_fp32, sizeof(float)*K*N) );
    CHECK_CUDA( hipMalloc(&dC_fp32, sizeof(float)*M*N) );

    CHECK_CUDA( hipMemcpy(dA_fp32, hA_fp32.data(), sizeof(float)*M*K, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB_fp32, hB_fp32.data(), sizeof(float)*K*N, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemset(dC_fp32, 0, sizeof(float)*M*N) );

    // 1) Run float32 GEMM
    float ms_fp32 = gemm_fp32(handle, M, N, K, dA_fp32, dB_fp32, dC_fp32, 50);
    float gfl_fp32 = calcGflops(M,N,K, ms_fp32);
    std::cout << "[FP32]  " << ms_fp32 << " ms, " << gfl_fp32 << " GFLOPS\n";

    // readback to host (if needed)
    CHECK_CUDA( hipMemcpy(hC_fp32.data(), dC_fp32, sizeof(float)*M*N, hipMemcpyDeviceToHost) );

    // 2) Run TF32 GEMM
    //    - same input arrays, but we set TF32 math mode
    CHECK_CUDA( hipMemset(dC_fp32, 0, sizeof(float)*M*N) );
    float ms_tf32 = gemm_tf32(handle, M, N, K, dA_fp32, dB_fp32, dC_fp32, 50);
    float gfl_tf32 = calcGflops(M,N,K, ms_tf32);
    std::cout << "[TF32]  " << ms_tf32 << " ms, " << gfl_tf32 << " GFLOPS\n";

    // 3) Run BF16 GEMM
    //    - we need __nv_bfloat16 buffers
    // create bf16 host buffers
    std::vector<__hip_bfloat16> hA_bf16(M*K), hB_bf16(K*N), hC_bf16(M*N);
    for(int i=0; i<M*K; i++){
        // float -> bfloat16
        hA_bf16[i] = __float2bfloat16(hA_fp32[i]);
    }
    for(int i=0; i<K*N; i++){
        hB_bf16[i] = __float2bfloat16(hB_fp32[i]);
    }
    for(int i=0; i<M*N; i++){
        hC_bf16[i] = __float2bfloat16(0.f);
    }

    // allocate device bf16
    __hip_bfloat16 *dA_bf16=nullptr, *dB_bf16=nullptr, *dC_bf16=nullptr;
    CHECK_CUDA( hipMalloc(&dA_bf16, sizeof(__hip_bfloat16)*M*K) );
    CHECK_CUDA( hipMalloc(&dB_bf16, sizeof(__hip_bfloat16)*K*N) );
    CHECK_CUDA( hipMalloc(&dC_bf16, sizeof(__hip_bfloat16)*M*N) );

    CHECK_CUDA( hipMemcpy(dA_bf16, hA_bf16.data(), sizeof(__hip_bfloat16)*M*K, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dB_bf16, hB_bf16.data(), sizeof(__hip_bfloat16)*K*N, hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dC_bf16, hC_bf16.data(), sizeof(__hip_bfloat16)*M*N, hipMemcpyHostToDevice) );

    float ms_bf16 = gemm_bf16(handle, M, N, K, dA_bf16, dB_bf16, dC_bf16, 50);
    float gfl_bf16 = calcGflops(M,N,K, ms_bf16);
    std::cout << "[BF16]  " << ms_bf16 << " ms, " << gfl_bf16 << " GFLOPS\n";

    // finalize
    CHECK_CUDA( hipFree(dA_fp32) );
    CHECK_CUDA( hipFree(dB_fp32) );
    CHECK_CUDA( hipFree(dC_fp32) );
    CHECK_CUDA( hipFree(dA_bf16) );
    CHECK_CUDA( hipFree(dB_bf16) );
    CHECK_CUDA( hipFree(dC_bf16) );

    hipblasDestroy(handle);
    return 0;
}

